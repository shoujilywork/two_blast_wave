#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdio.h>
#include "../include/split_lf.h"

#define BLOCK_SIZE 16

// CUDA核函数：计算声速c
__global__ void computeSoundSpeed(double* c, const double* r, const double* p, int rows, int cols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < rows && j < cols) {
        c[i*cols + j] = sqrt(fmax(0.0, 1.4 * p[i*cols + j] / r[i*cols + j]));
    }
}

// CUDA核函数：计算特征速度（并行归约优化版）
__global__ void computeCharacteristicSpeed(double* a, double* b, 
                                         const double* u, const double* v, 
                                         const double* c, int size) {
    extern __shared__ double sdata[];
    
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    double my_a = 0.0, my_b = 0.0;
    if (i < size) {
        my_a = fabs(u[i] + c[i]);
        my_b = fabs(v[i] + c[i]);
    }
    
    // 并行归约求最大值
    sdata[tid] = my_a;
    __syncthreads();
    
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmax(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        a[blockIdx.x] = sdata[0];
    }
    
    // 同样的方法计算b
    sdata[tid] = my_b;
    __syncthreads();
    
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmax(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        b[blockIdx.x] = sdata[0];
    }
}

// CUDA核函数：计算通量和分裂通量（使用共享内存优化）
__global__ void computeFluxes(double* fp1, double* fp2, double* fp3, double* fp4,
                             double* fn1, double* fn2, double* fn3, double* fn4,
                             double* gp1, double* gp2, double* gp3, double* gp4,
                             double* gn1, double* gn2, double* gn3, double* gn4,
                             const double* r, const double* p,
                             const double* u, const double* v,
                             const double* E, double a, double b,
                             int rows, int cols) {
    __shared__ double s_r[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double s_p[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double s_u[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double s_v[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double s_E[BLOCK_SIZE][BLOCK_SIZE];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int i = blockIdx.y * BLOCK_SIZE + ty;
    int j = blockIdx.x * BLOCK_SIZE + tx;
    
    // 加载数据到共享内存
    if (i < rows && j < cols) {
        s_r[ty][tx] = r[i*cols + j];
        s_p[ty][tx] = p[i*cols + j];
        s_u[ty][tx] = u[i*cols + j];
        s_v[ty][tx] = v[i*cols + j];
        s_E[ty][tx] = E[i*cols + j];
    }
    __syncthreads();
    
    if (i < rows && j < cols) {
        // 计算通量f和g
        double f1 = s_r[ty][tx] * s_u[ty][tx];
        double f2 = s_r[ty][tx] * s_u[ty][tx] * s_u[ty][tx] + s_p[ty][tx];
        double f3 = s_r[ty][tx] * s_u[ty][tx] * s_v[ty][tx];
        double f4 = (s_E[ty][tx] + s_p[ty][tx]) * s_u[ty][tx];
        
        double g1 = s_r[ty][tx] * s_v[ty][tx];
        double g2 = f3;
        double g3 = s_r[ty][tx] * s_v[ty][tx] * s_v[ty][tx] + s_p[ty][tx];
        double g4 = (s_E[ty][tx] + s_p[ty][tx]) * s_v[ty][tx];
        
        // 定义q变量
        double q1 = s_r[ty][tx];
        double q2 = f1;
        double q3 = g1;
        double q4 = s_E[ty][tx];
        
        // 计算分裂通量
        fp1[i*cols + j] = (f1 + a * q1) / 2;
        fn1[i*cols + j] = (f1 - a * q1) / 2;
        gp1[i*cols + j] = (g1 + b * q1) / 2;
        gn1[i*cols + j] = (g1 - b * q1) / 2;
        
        fp2[i*cols + j] = (f2 + a * q2) / 2;
        fn2[i*cols + j] = (f2 - a * q2) / 2;
        gp2[i*cols + j] = (g2 + b * q2) / 2;
        gn2[i*cols + j] = (g2 - b * q2) / 2;
        
        fp3[i*cols + j] = (f3 + a * q3) / 2;
        fn3[i*cols + j] = (f3 - a * q3) / 2;
        gp3[i*cols + j] = (g3 + b * q3) / 2;
        gn3[i*cols + j] = (g3 - b * q3) / 2;
        
        fp4[i*cols + j] = (f4 + a * q4) / 2;
        fn4[i*cols + j] = (f4 - a * q4) / 2;
        gp4[i*cols + j] = (g4 + b * q4) / 2;
        gn4[i*cols + j] = (g4 - b * q4) / 2;
    }
}

// 主函数：封装CUDA调用
void split_lf(const double* r, const double* p, const double* u, const double* v, const double* E,
                  double* fp1, double* fp2, double* fp3, double* fp4,
                  double* fn1, double* fn2, double* fn3, double* fn4,
                  double* gp1, double* gp2, double* gp3, double* gp4,
                  double* gn1, double* gn2, double* gn3, double* gn4,
                  int rows, int cols) {
    // 设备内存指针
    double *d_r, *d_p, *d_u, *d_v, *d_E;
    double *d_fp1, *d_fp2, *d_fp3, *d_fp4;
    double *d_fn1, *d_fn2, *d_fn3, *d_fn4;
    double *d_gp1, *d_gp2, *d_gp3, *d_gp4;
    double *d_gn1, *d_gn2, *d_gn3, *d_gn4;
    double *d_c, *d_a, *d_b;
    
    // 分配设备内存
    size_t size = rows * cols * sizeof(double);
    hipMalloc(&d_r, size);
    hipMalloc(&d_p, size);
    hipMalloc(&d_u, size);
    hipMalloc(&d_v, size);
    hipMalloc(&d_E, size);
    hipMalloc(&d_c, size);
    
    // 分配输出设备内存
    hipMalloc(&d_fp1, size);
    hipMalloc(&d_fp2, size);
    hipMalloc(&d_fp3, size);
    hipMalloc(&d_fp4, size);
    hipMalloc(&d_fn1, size);
    hipMalloc(&d_fn2, size);
    hipMalloc(&d_fn3, size);
    hipMalloc(&d_fn4, size);
    hipMalloc(&d_gp1, size);
    hipMalloc(&d_gp2, size);
    hipMalloc(&d_gp3, size);
    hipMalloc(&d_gp4, size);
    hipMalloc(&d_gn1, size);
    hipMalloc(&d_gn2, size);
    hipMalloc(&d_gn3, size);
    hipMalloc(&d_gn4, size);
    
    // 分配特征速度存储
    hipMalloc(&d_a, sizeof(double));
    hipMalloc(&d_b, sizeof(double));
    
    // 拷贝数据到设备
    hipMemcpy(d_r, r, size, hipMemcpyHostToDevice);
    hipMemcpy(d_p, p, size, hipMemcpyHostToDevice);
    hipMemcpy(d_u, u, size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, size, hipMemcpyHostToDevice);
    hipMemcpy(d_E, E, size, hipMemcpyHostToDevice);
    
    // 计算声速c
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x, 
                 (rows + blockDim.y - 1) / blockDim.y);
    computeSoundSpeed<<<gridDim, blockDim>>>(d_c, d_r, d_p, rows, cols);
    
    // 计算特征速度a和b
    int blockSize = 256;
    int gridSize = (rows * cols + blockSize - 1) / blockSize;
    computeCharacteristicSpeed<<<gridSize, blockSize, 2*blockSize*sizeof(double)>>>(
        d_a, d_b, d_u, d_v, d_c, rows * cols);
    
    // 主机上的a和b
    double h_a, h_b;
    hipMemcpy(&h_a, d_a, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&h_b, d_b, sizeof(double), hipMemcpyDeviceToHost);
    
    // 计算通量和分裂通量
    computeFluxes<<<gridDim, blockDim>>>(d_fp1, d_fp2, d_fp3, d_fp4,
                                       d_fn1, d_fn2, d_fn3, d_fn4,
                                       d_gp1, d_gp2, d_gp3, d_gp4,
                                       d_gn1, d_gn2, d_gn3, d_gn4,
                                       d_r, d_p, d_u, d_v, d_E, 
                                       h_a, h_b, rows, cols);
    
    // 拷贝结果回主机
    hipMemcpy(fp1, d_fp1, size, hipMemcpyDeviceToHost);
    hipMemcpy(fp2, d_fp2, size, hipMemcpyDeviceToHost);
    hipMemcpy(fp3, d_fp3, size, hipMemcpyDeviceToHost);
    hipMemcpy(fp4, d_fp4, size, hipMemcpyDeviceToHost);
    hipMemcpy(fn1, d_fn1, size, hipMemcpyDeviceToHost);
    hipMemcpy(fn2, d_fn2, size, hipMemcpyDeviceToHost);
    hipMemcpy(fn3, d_fn3, size, hipMemcpyDeviceToHost);
    hipMemcpy(fn4, d_fn4, size, hipMemcpyDeviceToHost);
    hipMemcpy(gp1, d_gp1, size, hipMemcpyDeviceToHost);
    hipMemcpy(gp2, d_gp2, size, hipMemcpyDeviceToHost);
    hipMemcpy(gp3, d_gp3, size, hipMemcpyDeviceToHost);
    hipMemcpy(gp4, d_gp4, size, hipMemcpyDeviceToHost);
    hipMemcpy(gn1, d_gn1, size, hipMemcpyDeviceToHost);
    hipMemcpy(gn2, d_gn2, size, hipMemcpyDeviceToHost);
    hipMemcpy(gn3, d_gn3, size, hipMemcpyDeviceToHost);
    hipMemcpy(gn4, d_gn4, size, hipMemcpyDeviceToHost);
    
    // 释放设备内存
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_E);
    hipFree(d_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_fp1);
    hipFree(d_fp2);
    hipFree(d_fp3);
    hipFree(d_fp4);
    hipFree(d_fn1);
    hipFree(d_fn2);
    hipFree(d_fn3);
    hipFree(d_fn4);
    hipFree(d_gp1);
    hipFree(d_gp2);
    hipFree(d_gp3);
    hipFree(d_gp4);
    hipFree(d_gn1);
    hipFree(d_gn2);
    hipFree(d_gn3);
    hipFree(d_gn4);
}