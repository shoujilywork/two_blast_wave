#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>

//
__global__ void max_reduce_kernel(const double* input, double* output, int size) {
    extern __shared__ double sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < size) ? input[i] : -INFINITY;
    __syncthreads();

    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmax(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) output[blockIdx.x] = sdata[0];
}

//
__global__ void split_lf_kernel(
    const double* r, const double* p, 
    const double* u, const double* v, const double* E,
    double* fp1, double* fp2, double* fp3, double* fp4,
    double* fn1, double* fn2, double* fn3, double* fn4,
    double* gp1, double* gp2, double* gp3, double* gp4,
    double* gn1, double* gn2, double* gn3, double* gn4,
    double a, double b, int rows, int cols)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < cols && j < rows) {
        int idx = j * cols + i;
        
        //
        double f1 = r[idx] * u[idx];
        double f2 = r[idx] * u[idx] * u[idx] + p[idx];
        double f3 = r[idx] * u[idx] * v[idx];
        double f4 = (E[idx] + p[idx]) * u[idx];
        
        double g1 = r[idx] * v[idx];
        double g2 = f3;
        double g3 = r[idx] * v[idx] * v[idx] + p[idx];
        double g4 = (E[idx] + p[idx]) * v[idx];
        
        // 
        double q1 = r[idx];
        double q2 = f1;
        double q3 = g1;
        double q4 = E[idx];
        
        // 
        fp1[idx] = (f1 + a * q1) / 2.0;
        fn1[idx] = (f1 - a * q1) / 2.0;
        gp1[idx] = (g1 + b * q1) / 2.0;
        gn1[idx] = (g1 - b * q1) / 2.0;
        
        fp2[idx] = (f2 + a * q2) / 2.0;
        fn2[idx] = (f2 - a * q2) / 2.0;
        gp2[idx] = (g2 + b * q2) / 2.0;
        gn2[idx] = (g2 - b * q2) / 2.0;
        
        fp3[idx] = (f3 + a * q3) / 2.0;
        fn3[idx] = (f3 - a * q3) / 2.0;
        gp3[idx] = (g3 + b * q3) / 2.0;
        gn3[idx] = (g3 - b * q3) / 2.0;
        
        fp4[idx] = (f4 + a * q4) / 2.0;
        fn4[idx] = (f4 - a * q4) / 2.0;
        gp4[idx] = (g4 + b * q4) / 2.0;
        gn4[idx] = (g4 - b * q4) / 2.0;
    }
}

void split_lf(
    const double* r, const double* p, 
    const double* u, const double* v, const double* E,
    double* fp1, double* fp2, double* fp3, double* fp4,
    double* fn1, double* fn2, double* fn3, double* fn4,
    double* gp1, double* gp2, double* gp3, double* gp4,
    double* gn1, double* gn2, double* gn3, double* gn4,
    int rows, int cols)
{
    // 1
    double *d_r, *d_p, *d_u, *d_v, *d_E;
    double *d_fp1, *d_fp2, *d_fp3, *d_fp4;
    double *d_fn1, *d_fn2, *d_fn3, *d_fn4;
    double *d_gp1, *d_gp2, *d_gp3, *d_gp4;
    double *d_gn1, *d_gn2, *d_gn3, *d_gn4;
    double *d_temp, *d_max;
    
    size_t size = rows * cols * sizeof(double);
    size_t temp_size = rows * cols;
    
    hipMalloc(&d_r, size);
    hipMalloc(&d_p, size);
    hipMalloc(&d_u, size);
    hipMalloc(&d_v, size);
    hipMalloc(&d_E, size);
    hipMalloc(&d_fp1, size);
    hipMalloc(&d_fp2, size);
    hipMalloc(&d_fp3, size);
    hipMalloc(&d_fp4, size);
    hipMalloc(&d_fn1, size);
    hipMalloc(&d_fn2, size);
    hipMalloc(&d_fn3, size);
    hipMalloc(&d_fn4, size);
    hipMalloc(&d_gp1, size);
    hipMalloc(&d_gp2, size);
    hipMalloc(&d_gp3, size);
    hipMalloc(&d_gp4, size);
    hipMalloc(&d_gn1, size);
    hipMalloc(&d_gn2, size);
    hipMalloc(&d_gn3, size);
    hipMalloc(&d_gn4, size);

    hipMalloc(&d_temp, temp_size * sizeof(double));
    hipMalloc(&d_max, sizeof(double));
    
    // 2. 
    hipMemcpy(d_r, r, size, hipMemcpyHostToDevice);
    hipMemcpy(d_p, p, size, hipMemcpyHostToDevice);
    hipMemcpy(d_u, u, size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, size, hipMemcpyHostToDevice);
    hipMemcpy(d_E, E, size, hipMemcpyHostToDevice);
    
    // 3.
    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, 
              (rows + block.y - 1) / block.y);

    double a = 0.0, b = 0.0; // 
    split_lf_kernel<<<grid, block>>>(
        d_r, d_p, d_u, d_v, d_E,
        d_fp1, d_fp2, d_fp3, d_fp4,
        d_fn1, d_fn2, d_fn3, d_fn4,
        d_gp1, d_gp2, d_gp3, d_gp4,
        d_gn1, d_gn2, d_gn3, d_gn4,
        a, b, rows, cols);
    
    // 5. 
    hipMemcpy(fp1, d_fp1, size, hipMemcpyDeviceToHost);
    hipMemcpy(fp2, d_fp2, size, hipMemcpyDeviceToHost);
    hipMemcpy(fp3, d_fp3, size, hipMemcpyDeviceToHost);
    hipMemcpy(fp4, d_fp4, size, hipMemcpyDeviceToHost);
    hipMemcpy(fn1, d_fn1, size, hipMemcpyDeviceToHost);
    hipMemcpy(fn2, d_fn2, size, hipMemcpyDeviceToHost);
    hipMemcpy(fn3, d_fn3, size, hipMemcpyDeviceToHost);
    hipMemcpy(fn4, d_fn4, size, hipMemcpyDeviceToHost);
    hipMemcpy(gp1, d_gp1, size, hipMemcpyDeviceToHost);
    hipMemcpy(gp2, d_gp2, size, hipMemcpyDeviceToHost);
    hipMemcpy(gp3, d_gp3, size, hipMemcpyDeviceToHost);
    hipMemcpy(gp4, d_gp4, size, hipMemcpyDeviceToHost);
    hipMemcpy(gn1, d_gn1, size, hipMemcpyDeviceToHost);
    hipMemcpy(gn2, d_gn2, size, hipMemcpyDeviceToHost);
    hipMemcpy(gn3, d_gn3, size, hipMemcpyDeviceToHost);
    hipMemcpy(gn4, d_gn4, size, hipMemcpyDeviceToHost);
    
    // 6
    hipFree(d_r); hipFree(d_p); hipFree(d_u); hipFree(d_v); hipFree(d_E);
    hipFree(d_fp1); hipFree(d_fp2); hipFree(d_fp3); hipFree(d_fp4);
    hipFree(d_fn1); hipFree(d_fn2); hipFree(d_fn3); hipFree(d_fn4);
    hipFree(d_gp1); hipFree(d_gp2); hipFree(d_gp3); hipFree(d_gp4);
    hipFree(d_gn1); hipFree(d_gn2); hipFree(d_gn3); hipFree(d_gn4);
    hipFree(d_temp); hipFree(d_max);
}
